// 
// Wrapper for CUDA functions
// 
// 2022, Jonathan Tainer
// 

#include <math.h>
#include "kernel.h"
#include "physics.h"

PointMass* createElements(int numOfElements) {
	PointMass* ptr;
	hipMalloc((void**)&ptr, sizeof(PointMass) * numOfElements);
	return ptr;
}

void deleteElements(PointMass* element) {
	hipFree(element);
}

void copyToDev(PointMass* devElement, PointMass* sysElement, int numOfElements) {
	hipMemcpy(devElement, sysElement, sizeof(PointMass) * numOfElements, hipMemcpyHostToDevice);
}

void copyToSys(PointMass* sysElement, PointMass* devElement, int numOfElements) {
	hipMemcpy(sysElement, devElement, sizeof(PointMass) * numOfElements, hipMemcpyDeviceToHost);
}

void step(PointMass* element, int numOfElements, float dt) {
	

	updateAcc<<<1, numOfElements>>>(element, numOfElements);
	updatePos<<<1, numOfElements>>>(element, numOfElements, dt);
}


